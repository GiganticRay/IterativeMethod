#include "CudaCheck.h"
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>
#include "mmio.h"
#include <string>

int main(){
	// load sparse matrix 
	std::string filePath = "/public/home/LeiChao/Document/IterativeMethod/DATA/1138_bus/1138_bus.mtx";
	FILE *f;
	MM_typecode matcode;
	int M, N, nz;
	int i, *I, *J;
	double *val;
	int ret_code;

	if ((f = fopen(filePath.c_str(), "r")) == NULL) 
        exit(1);
	
	// determine the type of matrix being represented in a Matrix Market file
    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        exit(1);
    }

	// get the basic info of sparse matrix
	if ((ret_code = mm_read_mtx_crd_size(f, &M, &N, &nz)) !=0)
        exit(1);

	I = (int *) malloc(nz * sizeof(int));
    J = (int *) malloc(nz * sizeof(int));
    val = (double *) malloc(nz * sizeof(double));

	for (i=0; i<nz; i++){
        fscanf(f, "%d %d %lg\n", &I[i], &J[i], &val[i]);
        I[i]--;  /* adjust from 1-based to 0-based */
        J[i]--;
    }

    if (f !=stdin) fclose(f);

	return 0;
}